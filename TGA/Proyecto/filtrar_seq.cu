#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <sys/resource.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

char *fileIN, *fileOUT;
unsigned char *image;
unsigned char *d_image; //device
int width, height, pixelWidth; //meta info de la imagen
int num_filtro;

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

#define BLANCO_Y_NEGRO 1
#define SATURAR 2
#define SEPIA 3
#define BRILLO 4
#define HISTOGRAMA 5

#define  Pr  .299
#define  Pg  .587
#define  Pb  .114



float saturacion; 
int brillo; //ESTÁ EN TANTO POR CIEN 

void Usage(){
	printf("Usage: ./exe fileIN fileOUT num_filtro [factor brillo o saturacion]\n");
	printf("num filtro:\n");
	printf("\t1: convertir a blanco y negro\n");
	printf("\t2: saturar imagen\n");
		printf("\t\tfactor saturacion: valor en coma flotante y en tanto por uno. Valor menor a 1 es quitar saturacion y mayor a 1 es aumentarla en el factor indicado\n");
		printf("\t\tSe recomienda usar como fileIN las imagenes IMG01.jpg e IMG02.jpg\n");
		printf("\t\tEjemplo de uso: ./filtrar.exe IMG01.jpg Out01.jpg 2 1.5\n");
	printf("\t3: filtro sepia\n");
	printf("\t4: cambiar brillo\n");
		printf("\t\tfactor brillo: valor entero en tanto por cien. Valor menor a 100 es quitar brillo y mayor a 100 es aumentarlo en el factor indicado\n");
		printf("\t\tEjemplo de uso: ./filtrar.exe IMG01.jpg Out01.jpg 2 150\n");
	exit(0);
}

// Retorna un 0 si el maximo es 'a', un 1 si es 'b' y un 2 si es 'c'
int maximo(int a, int b, int c){
	if      (a >= b && a >= c) return 0;
	else if (b >= a && b >= c) return 1;
	else return 2;
}

float GetTime(void); 


int main(int argc, char** argv)
{
  float t1,t2, TiempoSEQ;

  // Ficheros de entrada y de salida 
  if (argc == 4) {
	fileIN = argv[1]; fileOUT = argv[2]; num_filtro = atoi(argv[3]);
	if (num_filtro == SATURAR || num_filtro == BRILLO) Usage();
  }
  else if (argc == 5) {
	fileIN = argv[1]; fileOUT = argv[2]; num_filtro = atoi(argv[3]); brillo = atoi(argv[4]); saturacion = atof(argv[4]);
	if (num_filtro == SATURAR || num_filtro == BRILLO){
		if (brillo < 0 || saturacion < 0) Usage();
	} 
	else Usage();
  }
  else {
	Usage();
  }

  printf("Reading image...\n");
  image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);
  if (!image) {
    fprintf(stderr, "Couldn't load image.\n");
     return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);

  if(num_filtro == BLANCO_Y_NEGRO){
	  t1=GetTime();
	  int C; int size = width*height*3;
	  for(int i=0; i<size; i=i+3){
		  C = 299 * image[i];
		  C += 587 * image[i+1];
		  C += 114 * image[i+2];
		  image[i] = image[i+1] = image[i+2] = C/1000;
	  }
	  t2=GetTime();
	  TiempoSEQ = (t2 - t1);
  }
  else if (num_filtro == SATURAR){
	t1=GetTime();
	int size = width*height*3; int R, G, B; double P;
	for(int i=0; i<size; i=i+3){
		R = image[i]; G = image[i+1]; B = image[i+2];
		P = sqrt( R*R*Pr + G*G*Pg + B*B*Pb ) ;
		R= MIN(P+(R-P)*saturacion , 255);
		G= MIN(P+(G-P)*saturacion , 255);
		B= MIN(P+(B-P)*saturacion , 255);
		image[i] = R; image[i+1] = G; image[i+2] = B;
	}
	t2=GetTime();
	TiempoSEQ = (t2 - t1);
  }
  else if (num_filtro == SEPIA){
	//https://abhijitnathwani.github.io/blog/2018/01/08/colortosepia-Image-using-C
	t1=GetTime();
	int size = width*height*3; int R, G, B;
	for(int i=0; i<size; i=i+3){
		R = MIN(255, (image[i]*0.393) + (image[i+1]*0.769) + (image[i+2]*0.189));
		G = MIN(255, (image[i]*0.349) + (image[i+1]*0.686) + (image[i+2]*0.168));
		B = MIN(255, (image[i]*0.272) + (image[i+1]*0.534) + (image[i+2]*0.131));
		image[i] = R; image[i+1] = G; image[i+2] = B;
	}
	t2=GetTime();
	TiempoSEQ = (t2 - t1);
  } 
  else if (num_filtro == BRILLO){
	t1=GetTime();
	int size = width*height*3; int R, G, B;
	for(int i=0; i<size; i=i+3){
		R = MIN(255, (image[i]*brillo/100));
		G = MIN(255, (image[i+1]*brillo/100));
		B = MIN(255, (image[i+2]*brillo)/100);
		image[i] = R; image[i+1] = G; image[i+2] = B;
	}
	t2=GetTime();
	TiempoSEQ = (t2 - t1);
  }  
  else if (num_filtro == HISTOGRAMA) {
	//t1=GetTime();
	int redH[256] = {0}; 
	int greenH[256] = {0}; 
	int blueH[256] = {0}; 
	int size = width*height*3; 
	
	for (int i=0; i<size; i=i+3) {
		redH[image[i]]++;
		greenH[image[i+1]]++;
	blueH[image[i+2]]++;
	}

	int minR, minG, minB, maxR, maxG, maxB;
	bool found = false;
	for (int i=0; found==false && i<256; ++i) {
		if (redH[i]>3) {
			minR=i;
			found = true;
		}
	}
	found = false;
	for (int i=0; found==false && i<256; ++i) {
		if (greenH[i]>3) {
			minG=i;
			found = true;
		}
	}
	found = false;
	for (int i=0; found==false && i<256; ++i) {
		if (blueH[i]>3) {
			minB=i;
			found = true;
		}
	}
	found = false;
	for (int i=255; found==false && i>0; --i) {
		if (redH[i]>3) {
			maxR=i;
			found = true;
		}
	}
	found = false;
	for (int i=255; found==false && i>0; --i) {
		if (greenH[i]>3) {
			maxG=i;
			found = true;
		}
	}
	found = false;
	for (int i=255; found==false && i>0; --i) {
		if (blueH[i]>3) {
			maxB=i;
			found = true;
		}
	}
	unsigned char* redChannel = (unsigned char*)malloc(height*width);
	unsigned char* greenChannel = (unsigned char*)malloc(height*width);
	unsigned char* blueChannel = (unsigned char*)malloc(height*width);
	for (int i=0; i<height*width; i++) {
		redChannel[i] = image[i*3];
		greenChannel[i] = image[i*3+1];
		blueChannel[i] = image[i*3+2];
	}
	int R,G,B;
	double multR = (double)255/((double)maxR-(double)minR);
	double multG = (double)255/((double)maxG-(double)minG);
	double multB = (double)255/((double)maxB-(double)minB);
	t1=GetTime();	
	for (int i=0; i<height*width; ++i) {
		R = (redChannel[i]-minR)*multR;
		image[i*3] = MIN(255, R);
		G = (greenChannel[i]-minG)*multG;
		image[i*3+1] = MIN(255, G);
		B = (blueChannel[i]-minB)*multB;
		image[i*3+2] = MIN(255, B);
	}
	t2=GetTime();
	TiempoSEQ = (t2 - t1);
	free(redChannel);
	free(greenChannel);
	free(blueChannel);

	
  }


  printf("TIEMPO SECUENCIAL: %4.6f milseg\n", TiempoSEQ);
  
    
  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,image,0);
  

}


float GetTime(void) {
  struct timeval tim;
  struct rusage ru;
  getrusage(RUSAGE_SELF, &ru);
  tim=ru.ru_utime;
  return ((double)tim.tv_sec + (double)tim.tv_usec / 1000000.0)*1000.0;
}

