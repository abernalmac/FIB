#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

char *fileIN, *fileOUT;
unsigned char *image;
unsigned char *d_image; //device
unsigned char *redChannel;
unsigned char *d_redChannel; //device
unsigned char *greenChannel;
unsigned char *d_greenChannel; //device
unsigned char *blueChannel;
unsigned char *d_blueChannel; //device
int width, height, pixelWidth; //meta info de la imagen
int num_filtro;
float saturacion; 
int brillo;

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

#define BLANCO_Y_NEGRO 1
#define SATURAR 2
#define SEPIA 3
#define BRILLO 4
#define HISTOGRAMA 5

void Usage(){
	printf("Usage: ./exe fileIN fileOUT num_filtro [factor brillo o saturacion]\n");
	printf("num filtro:\n");
	printf("\t1: convertir a blanco y negro\n");
	printf("\t2: saturar imagen\n");
		printf("\t\tfactor saturacion: valor en coma flotante y en tanto por uno. Valor menor a 1 es quitar saturacion y mayor a 1 es aumentarla en el factor indicado\n");
		printf("\t\tSe recomienda usar como fileIN las imagenes IMG01.jpg e IMG02.jpg\n");
		printf("\t\tEjemplo de uso: ./filtrar.exe IMG01.jpg Out01.jpg 2 1.5\n");
	printf("\t3: filtro sepia\n");
	printf("\t4: cambiar brillo\n");
		printf("\t\tfactor brillo: valor entero en tanto por cien. Valor menor a 100 es quitar brillo y mayor a 100 es aumentarlo en el factor indicado\n");
		printf("\t\tEjemplo de uso: ./filtrar.exe IMG01.jpg Out01.jpg 2 150\n");
	exit(0);
}

void CheckCudaError(char sms[], int line);

__global__ void KernelBlancoNegro_UsingFloats(unsigned char *image, int N) {
  int elem = (blockIdx.x * blockDim.x + threadIdx.x)*3; //cada thread se ocupa de 3 posiciones de imagen (RGB de 1 pixel)
  if(elem < N){
	float C;
	C = 0.299 * image[elem];
        C += 0.587 * image[elem+1];
        C += 0.114 * image[elem+2];
        image[elem] = image[elem+1] = image[elem+2] = C;
  }
}

__global__ void KernelBlancoNegro(unsigned char *image, int N) {
  int elem = (blockIdx.x * blockDim.x + threadIdx.x)*3; //cada thread se ocupa de 3 posiciones de imagen (RGB de 1 pixel)
  if(elem < N){
	int C;
	C = 299 * image[elem];
        C += 587 * image[elem+1];
        C += 114 * image[elem+2];
        image[elem] = image[elem+1] = image[elem+2] = C/1000;
  }
}

__global__ void KernelSaturar(unsigned char *image, float saturacion, int N) {
  int elem = (blockIdx.x * blockDim.x + threadIdx.x)*3; 
  if(elem < N){
	int R, G, B; float P;
        R = image[elem]; G = image[elem+1]; B = image[elem+2];
        P = sqrt( R*R*0.299 + G*G*0.587 + B*B*0.114 );
        R= MIN(P+(R-P)*saturacion , 255);
        G= MIN(P+(G-P)*saturacion , 255);
        B= MIN(P+(B-P)*saturacion , 255);
        image[elem] = R; image[elem+1] = G; image[elem+2] = B;
  }
}

__global__ void KernelSepia(unsigned char *image, int N) {
  int elem = (blockIdx.x * blockDim.x + threadIdx.x)*3; 
  if(elem < N){
	int R, G, B;
    R = MIN(255, (image[elem]*393 + image[elem+1]*769 + image[elem+2]*189)/1000 );
	G = MIN(255, (image[elem]*349 + image[elem+1]*686 + image[elem+2]*168)/1000 );
	B = MIN(255, (image[elem]*272 + image[elem+1]*534 + image[elem+2]*131)/1000 );
	image[elem] = R; image[elem+1] = G; image[elem+2] = B;
  }
}

__global__ void KernelBrillo(unsigned char *image, float brillo, int N) {
  int elem = (blockIdx.x * blockDim.x + threadIdx.x)*3; 
  if(elem < N){
	int R, G, B;
	R = MIN(255, (image[elem]*brillo/100));
	G = MIN(255, (image[elem+1]*brillo/100));
	B = MIN(255, (image[elem+2]*brillo)/100);
	image[elem] = R; image[elem+1] = G; image[elem+2] = B; 
  }
}

__global__ void KernelHistograma(unsigned char *image, unsigned char *redChannel, unsigned char *greenChannel, unsigned char *blueChannel, int minR, int minG, int minB, double multR, double multG, double multB, int N) {
	int elem = (blockIdx.x * blockDim.x + threadIdx.x);
	if (elem < N) {
		int R, G, B;
		R = (redChannel[elem]-minR)*multR;
		image[elem*3] = MIN(255, R);
		G = (greenChannel[elem]-minG)*multG;
		image[elem*3+1] = MIN(255, G);
		B = (blueChannel[elem]-minB)*multB;
		image[elem*3+2] = MIN(255, B);
	}
}

int main(int argc, char** argv)
{
   
  unsigned int numBytes, numPixels;
  unsigned int nBlocks, nThreads;
  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  // Ficheros de entrada y de salida 
  if (argc == 4) {
	fileIN = argv[1]; fileOUT = argv[2]; num_filtro = atoi(argv[3]);
	if (num_filtro == SATURAR || num_filtro == BRILLO) Usage();
  }
  else if (argc == 5) {
	fileIN = argv[1]; fileOUT = argv[2]; num_filtro = atoi(argv[3]); brillo = atoi(argv[4]); saturacion = atof(argv[4]);
	if (num_filtro == SATURAR || num_filtro == BRILLO){
		if (brillo < 0 || saturacion < 0) Usage();
	} 
	else Usage();
  }
  else {
	Usage();
  }

  printf("Reading image...\n");
  image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);
  if (!image) {
    fprintf(stderr, "Couldn't load image.\n");
     return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);

  int count, gpu;
  // Buscar GPU de forma aleatoria. 
  hipGetDeviceCount(&count);
  srand(time(NULL));
  gpu = (rand()>>3) % count;
  hipSetDevice(gpu);

  // numero de Threads, bloques, bytes:
  nThreads = 1024;
  numPixels = width * height;
  numBytes = numPixels * pixelWidth; //pixelWidth=3
  nBlocks = (numPixels+nThreads-1)/nThreads;

  //creamos eventos
  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  // guardamos tiempos previos a coger espacio en GPU y pasarle datos
  hipEventRecord(E0, 0);
  hipEventSynchronize(E0); 

  // Obtener Memoria en el device
  hipMalloc((unsigned char**)&d_image, numBytes);
  CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);

  // Copiar datos desde el host en el device 
  hipMemcpy(d_image, image, numBytes, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

  // guardamos tiempos posteriores a gestion de datos y previos al kernel
  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);


  //ejecucion kernel:
  if(num_filtro == BLANCO_Y_NEGRO){
	KernelBlancoNegro<<<nBlocks, nThreads>>>(d_image, numBytes); 
  } else if (num_filtro == SATURAR){
	KernelSaturar<<<nBlocks, nThreads>>>(d_image, saturacion, numBytes);
  } else if (num_filtro == SEPIA){
	KernelSepia<<<nBlocks, nThreads>>>(d_image, numBytes);
  } else if (num_filtro == BRILLO){
	KernelBrillo<<<nBlocks, nThreads>>>(d_image, brillo, numBytes);
  } else if (num_filtro == HISTOGRAMA){
	int redH[256] = {0}; 
	int greenH[256] = {0}; 
	int blueH[256] = {0}; 
	int size = width*height*3; 
	for (int i=0; i<size; i=i+3) {
		redH[image[i]]++;
		greenH[image[i+1]]++;
		blueH[image[i+2]]++;
	}
	int minR, minG, minB, maxR, maxG, maxB;
	bool found = false;
	for (int i=0; found==false && i<256; ++i) {
		if (redH[i]>3) {
			minR=i;
			found = true;
		}
	}
	found = false;
	for (int i=0; found==false && i<256; ++i) {
		if (greenH[i]>3) {
			minG=i;
			found = true;
		}
	}
	found = false;
	for (int i=0; found==false && i<256; ++i) {
		if (blueH[i]>3) {
			minB=i;
			found = true;
		}
	}
	found = false;
	for (int i=255; found==false && i>0; --i) {
		if (redH[i]>3) {
			maxR=i;
			found = true;
		}
	}
	found = false;
	for (int i=255; found==false && i>0; --i) {
		if (greenH[i]>3) {
			maxG=i;
			found = true;
		}
	}
	found = false;
	for (int i=255; found==false && i>0; --i) {
		if (blueH[i]>3) {
			maxB=i;
			found = true;
		}
	}
	redChannel = (unsigned char*)malloc(height*width);
	greenChannel = (unsigned char*)malloc(height*width);
	blueChannel = (unsigned char*)malloc(height*width);
	for (int i=0; i<height*width; i++) {
		redChannel[i] = image[i*3];
		greenChannel[i] = image[i*3+1];
		blueChannel[i] = image[i*3+2];
	}

	double multR = (double)255/((double)maxR-(double)minR);
	double multG = (double)255/((double)maxG-(double)minG);
	double multB = (double)255/((double)maxB-(double)minB);

	hipMalloc((unsigned char**)&d_redChannel, numPixels);
	CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
	hipMemcpy(d_redChannel, redChannel, numPixels, hipMemcpyHostToDevice);
	CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
	
	hipMalloc((unsigned char**)&d_greenChannel, numPixels);
	CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
	hipMemcpy(d_greenChannel, greenChannel, numPixels, hipMemcpyHostToDevice);
	CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
	
	hipMalloc((unsigned char**)&d_blueChannel, numPixels);
	CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
	hipMemcpy(d_blueChannel, blueChannel, numPixels, hipMemcpyHostToDevice);
	CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

	KernelHistograma<<<nBlocks, nThreads>>>(d_image, d_redChannel, d_greenChannel, d_blueChannel, minR, minG, minB, multR, multG, multB, numPixels);
	CheckCudaError((char *) "Invocar Kernel", __LINE__);

	hipFree(d_redChannel);
	hipFree(d_greenChannel);
	hipFree(d_blueChannel);
  }
  //miramos errores invocacion de los kernels ByN, Sat, Sepia, Brillo.
  CheckCudaError((char *) "Invocar Kernel", __LINE__);

  // guardamos tiempos posteriores al kernel y previos a la copia D->H
  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host 
  hipMemcpy(image, d_image, numBytes, hipMemcpyDeviceToHost);
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

  // Liberar Memoria del device 
  hipFree(d_image); 

  // guardamos tiempos finales 
  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  // tiempos
  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);

  // reporte
  printf("\nREPORTE FINAL:\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);

  //destruccion eventos
  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

    
  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,image,0);

}

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
  //else printf("(OK) %s \n", sms);
}

